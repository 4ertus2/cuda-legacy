
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#define cudaErrCheck(code) if (code != hipSuccess) throw CudaException{code, __FILE__, __LINE__}

struct CudaException
{
    hipError_t code;
    const char * file;
    int line;
    
    const char * what() const noexcept { return hipGetErrorString(code); }
};

__global__ void kernel(int * x, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size)
        return;
    x[id] = id;
}

int main()
{
    try
    {
        size_t size = 1024;
        std::vector<int> x(size);

        int * dev_x = 0;
        cudaErrCheck(hipMalloc((void **) &dev_x, size * sizeof(int)));
        cudaErrCheck(hipMemcpy(dev_x, &x[0], size * sizeof(int), hipMemcpyHostToDevice));

        size_t num_blocks = 16;
        size_t threads_per_block = 64;
        
        kernel<<<num_blocks, threads_per_block>>>(dev_x, size);
        cudaErrCheck(hipPeekAtLastError());
        cudaErrCheck(hipDeviceSynchronize());
        
        cudaErrCheck(hipMemcpy(&x[0], dev_x, size * sizeof(int), hipMemcpyDeviceToHost));
        cudaErrCheck(hipFree(dev_x));
        
        for (size_t i = 0; i < size; ++i)
            if (i != x[i])
                std::cout << "not OK at position: " << i << std::endl;
        std::cout << "done" << std::endl;
    }
    catch (const CudaException & ex)
    {
        std::cerr << ex.file << ":" << ex.line << " " << ex.what() << std::endl;
        return 1;
    }

    return 0;
}
